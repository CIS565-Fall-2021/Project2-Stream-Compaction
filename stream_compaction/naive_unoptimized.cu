#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 1024
dim3 threadsPerBlock(blockSize);

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        int* dev_array;
        
        __global__ void kernScanLayer(
            int array_length, int stride, int* array) {
            // compute one layer of scan in parallel.
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= array_length - stride) {
                return;
            }
            array[index + stride] = array[index] + array[index + stride];
            __syncthreads();
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // deal with non-2-power input
            int depth = ilog2ceil(n);
            int array_length = pow(2, depth);
            if (ilog2(n) != depth) {
                int* new_idata = new int[array_length];
                memset(new_idata, 0, array_length * sizeof(int));
                memcpy(new_idata, idata, n * sizeof(int));
                idata = new_idata;
            }

            dim3 fullBlocksPerGrid((array_length + blockSize - 1) / blockSize);
            hipMalloc((void**)&dev_array, array_length * sizeof(int));
            hipMemcpy(dev_array + 1, idata, (array_length - 1) * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(dev_array, 0, 1);

            timer().startGpuTimer();
            for (int depth_ind = 1; depth_ind <= depth; depth_ind++) {
                int stride = pow(2, depth_ind - 1);
                kernScanLayer << <fullBlocksPerGrid, blockSize >> > (array_length, stride, dev_array);
                
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_array, array_length * sizeof(int), hipMemcpyDeviceToHost);

            //int* array_0 = new int[array_length];
            //int* array_1 = new int[array_length];
            //hipMemcpy(array_0, dev_array_dep1, array_length * sizeof(int), hipMemcpyDeviceToHost);
            //hipMemcpy(array_1, dev_array_dep2, array_length * sizeof(int), hipMemcpyDeviceToHost);
            //printf("\n");
            //printf("\n");
            //for (int ind = 0; ind < array_length; ind++) {
            //    printf("%d ", array_0[ind]);
            //}
            //printf("\n");
            //printf("\n");
            //for (int ind = 0; ind < array_length; ind++) {
            //    printf("%d ", array_1[ind]);
            //}
            //printf("\n");
            //printf("\n");
            
            
        }
    }
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 1024
//dim3 threadsPerBlock(blockSize);

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        int* dev_array;

        //__global__ void kernReduction_1st_attempt(
        //    int array_length, int sum_ind_diff, int start_ind, int stride,
        //    int* array) {
        //    // compute one layer of scan in parallel.
        //    int index = threadIdx.x + (blockIdx.x * blockDim.x);
        //    if (index * stride + sum_ind_diff + start_ind >= array_length) {
        //        return;
        //    }
        //    array[index * stride + sum_ind_diff + start_ind] = array[index * stride + start_ind] + array[index * stride + sum_ind_diff + start_ind];
        //    __syncthreads();
        //}

        //__global__ void kernScanFromReduction_1st_attempt(
        //    int array_length, int sum_ind_diff, int start_ind, int stride,
        //    int* array) {
        //    int index = threadIdx.x + (blockIdx.x * blockDim.x);
        //    if (array_length-1 - index * stride - sum_ind_diff - start_ind < 0) {
        //        return;
        //    }
        //    int left_child = array[array_length - 1 - index * stride - sum_ind_diff];
        //    array[array_length - 1 - index * stride - sum_ind_diff] = array[array_length - 1 - index * stride];
        //    array[array_length - 1 - index * stride] = array[array_length - 1 - index * stride] + left_child;

        //}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        //void scan_1st_attempt(int n, int* odata, const int* idata) {
        //    // deal with non-2-power input
        //    int depth = ilog2ceil(n);
        //    int array_length = pow(2, depth);
        //    if (ilog2(n) != depth) {
        //        int* new_idata = new int[array_length];
        //        memset(new_idata, 0, array_length * sizeof(int));
        //        memcpy(new_idata, idata, n * sizeof(int));
        //        idata = new_idata;
        //    }
        //    hipMalloc((void**)&dev_array, array_length * sizeof(int));
        //    hipMemcpy(dev_array, idata, array_length * sizeof(int), hipMemcpyHostToDevice);

        //    timer().startGpuTimer();
        //    dim3 fullBlocksPerGrid((array_length + blockSize - 1) / blockSize);
        //    for (int depth_ind = 0; depth_ind <= depth - 1; depth_ind++) {
        //        int sum_ind_diff = pow(2, depth_ind);
        //        int start_ind = sum_ind_diff - 1;
        //        int stride = pow(2, depth_ind + 1);
        //        kernReduction_1st_attempt << <fullBlocksPerGrid, blockSize >> > (array_length, sum_ind_diff, start_ind, stride, dev_array);
        //    }
        //    hipDeviceSynchronize();

        //    hipMemset(dev_array + array_length - 1, 0, sizeof(int));
        //    for (int depth_ind = depth - 1; depth_ind >=0 ; depth_ind--) {
        //        int sum_ind_diff = pow(2, depth_ind);
        //        int start_ind = sum_ind_diff - 1;
        //        int stride = pow(2, depth_ind + 1);
        //        kernScanFromReduction_1st_attempt << <fullBlocksPerGrid, blockSize >> > (array_length, sum_ind_diff, start_ind, stride, dev_array);
        //    }
        //    timer().endGpuTimer();
        //    hipMemcpy(odata, dev_array, array_length * sizeof(int), hipMemcpyDeviceToHost);

        //    //for (int ind = 0; ind < array_length; ind++) {
        //    //    printf("%d ", odata[ind]);
        //    //}
        //    //printf("\n");
        //    //printf("\n");
        //}

        __global__ void kernReduction(
            int array_length, int* array) {
            // compute scan in parallel.
            __shared__ int share_array[blockSize];
            int tx = threadIdx.x;
            if (tx >= array_length) {
                return;
            }
            share_array[tx] = array[tx];
            __syncthreads();
            for (int stride = 1; stride < blockDim.x; stride *= 2) {
                if (tx % (2 * stride) == (2 * stride) - 1) {
                    share_array[tx] += share_array[tx - stride];
                }
                __syncthreads();
            }
            array[tx] = share_array[tx];
        }

        //__global__ void kernReduction(
        //    int array_length, int* array) {
        //    // compute one layer of scan in parallel.
        //    int tx = threadIdx.x;
        //    if (tx >= array_length) {
        //        return;
        //    }
        //    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        //        if (tx % (2 * stride) == (2 * stride) - 1) {
        //            array[tx] += array[tx - stride];
        //        }
        //        __syncthreads();
        //    }
        //}

        //__global__ void kernScanFromReduction(
        //    int array_length, int depth, int* array) {
        //    int tx = threadIdx.x;
        //    if (tx >= array_length) {
        //        return;
        //    }
        //    for (int depth_ind = depth; depth_ind > 0; depth_ind--) {
        //        if (tx % (2 * stride) == (2 * stride) - 1) {
        //            array[tx] += array[tx - stride];
        //        }
        //        __syncthreads();
        //    }
        //}

        void scan(int n, int* odata, const int* idata) {
            int depth = ilog2ceil(n);
            int array_length = pow(2, depth);
            if (ilog2(n) != depth) {
                int* new_idata = new int[array_length];
                memset(new_idata, 0, array_length * sizeof(int));
                memcpy(new_idata, idata, n * sizeof(int));
                idata = new_idata;
            }
            hipMalloc((void**)&dev_array, array_length * sizeof(int));
            hipMemcpy(dev_array, idata, array_length * sizeof(int), hipMemcpyHostToDevice);
            dim3 fullBlocksPerGrid((array_length + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            kernReduction << <fullBlocksPerGrid, blockSize >> > (array_length, dev_array);
            hipDeviceSynchronize();

            printf("%d ", depth);
            printf("\n");

            //hipMemset(dev_array + array_length - 1, 0, sizeof(int));
            //kernScanFromReduction << <fullBlocksPerGrid, blockSize >> > (array_length, depth, dev_array);
            timer().endGpuTimer();
            //hipMemcpy(odata, dev_array, array_length * sizeof(int), hipMemcpyDeviceToHost);

            for (int ind = 0; ind < array_length; ind++) {
                printf("%d ", odata[ind]);
            }
            printf("\n");
            printf("\n");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}

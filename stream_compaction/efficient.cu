#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction
{
    namespace Efficient
    {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer &timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernScanEfficientUpSweep(int n, int layer, int shift, int *data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n)
            {
                return;
            }
            int validFlag = !((index + 1) & ((1 << (layer + 1)) - 1));
            // look at the right place, multiply by whether the bottom bits all 1
            int otherVal = validFlag * data[index - shift];
            __syncthreads();
            data[index] += otherVal;
        }

        __global__ void kernScanEfficientDownSweep(int n, int layer, int max, int shift, int *data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n)
            {
                return;
            }
            // int rChildFlag = (((index + 1) & ((1 << (layer + 1)) - 1)) == 0 ? 1 : 0);
            // int lChildFlag = !rChildFlag &&
            //                  (((index + 1) & ((1 << (layer)) - 1)) == 0 ? 1 : 0);
            int rChildFlag = !((index + 1) & ((1 << (layer + 1)) - 1));
            int lChildFlag = !rChildFlag &&
                             !((index + 1) & ((1 << (layer)) - 1));
            int nextVal = !lChildFlag * (data[index] + (rChildFlag * data[index - shift])) +
                          lChildFlag * data[index + shift];
            // nextVal *= !((index == n - 1) && (layer == max));
            __syncthreads();
            data[index] = nextVal;
        }

        __global__ void kernSetLastToZero(int n, int *data)
        {
            data[n - 1] = 0;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata)
        {
            //next power of 2
            int power = ilog2ceil(n);
            int size = pow(2, power);
            int offset = size - n;
            dim3 fullBlocksPerGrid((size + blockSize - 1) / blockSize);
            int *buf;
            hipMalloc((void **)&buf, size * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc buf failed!");
            hipMemset(buf, 0, size * sizeof(int));
            checkCUDAErrorWithLine("hipMemset buf failed!");
            hipMemcpy(buf + offset, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("hipMemcpy buf failed!");
            timer().startGpuTimer();
            // TODO
            for (int layer = 0; layer < power; layer++)
            {
                // invoke kernel
                int shift = pow(2, layer);
                kernScanEfficientUpSweep<<<fullBlocksPerGrid, blockSize>>>(size, layer, shift, buf);
                hipDeviceSynchronize();
                // checkCUDAErrorWithLine("hipDeviceSynchronize buf failed!");
            }
            kernSetLastToZero<<<1, 1>>>(size, buf);
            hipDeviceSynchronize();
            for (int layer = power - 1; layer >= 0; layer--)
            {
                // invoke kernel
                int shift = pow(2, layer);
                kernScanEfficientDownSweep<<<fullBlocksPerGrid, blockSize>>>(size, layer, power - 1, shift, buf);
                hipDeviceSynchronize();
                // checkCUDAErrorWithLine("hipDeviceSynchronize buf failed!");
            }
            timer().endGpuTimer();
            hipMemcpy(odata, buf + offset, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorWithLine("hipMemcpy buf failed!");
            hipFree(buf);
            checkCUDAErrorWithLine("hipFree buf failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata)
        {
            //next power of 2
            int power = ilog2ceil(n);
            int size = pow(2, power);
            int offset = size - n;
            dim3 fullBlocksPerGrid((size + blockSize - 1) / blockSize);

            int *buf; // power of 2 0 padded copy of idata
            hipMalloc((void **)&buf, size * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc buf failed!");
            hipMemset(buf, 0, size * sizeof(int));
            checkCUDAErrorWithLine("hipMemset buf failed!");
            hipMemcpy(buf + offset, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("hipMemcpy buf failed!");

            int *bools, *indices, *tmpOut; //
            hipMalloc((void **)&bools, size * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc bools failed!");
            hipMalloc((void **)&indices, size * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc indices failed!");
            hipMalloc((void **)&tmpOut, size * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc tmpOut failed!");
            timer().startGpuTimer();
            // TODO
            // Map
            Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(size, bools, buf);
            Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(size, indices, buf);
            hipDeviceSynchronize();
            checkCUDAErrorWithLine("hipDeviceSynchronize failed!");
            // Scan
            for (int layer = 0; layer < power; layer++)
            {
                // invoke kernel
                int shift = pow(2, layer);
                kernScanEfficientUpSweep<<<fullBlocksPerGrid, blockSize>>>(size, layer, shift, indices);
                hipDeviceSynchronize();
                checkCUDAErrorWithLine("hipDeviceSynchronize failed!");
            }
            kernSetLastToZero<<<1, 1>>>(size, indices);
            hipDeviceSynchronize();
            checkCUDAErrorWithLine("hipDeviceSynchronize failed!");
            for (int layer = power - 1; layer >= 0; layer--)
            {
                // invoke kernel
                int shift = pow(2, layer);
                kernScanEfficientDownSweep<<<fullBlocksPerGrid, blockSize>>>(size, layer, power - 1, shift, indices);
                hipDeviceSynchronize();
                checkCUDAErrorWithLine("hipDeviceSynchronize failed!");
            }
            // Scatter
            Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(size, tmpOut, buf, bools, indices);
            hipDeviceSynchronize();
            checkCUDAErrorWithLine("hipDeviceSynchronize failed!");
            timer().endGpuTimer();

            hipMemcpy(odata, tmpOut + offset, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorWithLine("hipMemcpy failed!");
            hipFree(buf);
            checkCUDAErrorWithLine("hipFree buf failed!");
            hipFree(bools);
            checkCUDAErrorWithLine("hipFree bools failed!");
            hipFree(indices);
            checkCUDAErrorWithLine("hipFree indices failed!");
            hipFree(tmpOut);
            checkCUDAErrorWithLine("hipFree tmpOut failed!");
            return -1;
        }
    }
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include "cVec.h"

#define blockSize 4

namespace StreamCompaction {
namespace Efficient {
	using StreamCompaction::Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}


	template <typename T>
	__global__ void kern_up_sweep(int d, int n, T *x)
	{
		int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
		int k = idx * (1 << (d+1));
		if (k >= n)
			return;
		x[k + (1<<(d+1)) - 1] = x[k + (1<<(d+1)) - 1] + x[k + (1<<d) - 1];
	}


	template <typename T>
	__global__ void kern_up_sweep2(int n, T* x)
	{
		int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

		for (int stride = 1; stride < n; stride *= 2) {

			int src = idx * stride * 2 + stride - 1;
			int dst = src + stride;

			if (dst >= n)
				return;

			x[dst] += x[src];
			__syncthreads();
		}
	}


	template <typename T>
	__global__ void kern_down_sweep(int d, int n, T *x)
	{
		int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
		int k = idx * (1 << (d+1));

		int t = x[k+(1<<d)-1];
		
		x[k+(1<<d)-1] = x[k+(1<<(d+1))-1];
		x[k+(1<<(d+1))-1] = t + x[k+(1<<(d+1))-1];
	}

	template <typename T>
	__global__ void kern_down_sweep2(int n, T* x)
	{
		int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (idx >= n)
			return;

		for (int stride = n/2; stride >= 1; stride /= 2) {

			int src = idx * stride * 2 + stride - 1;
			int dst = src + stride;
	
			if (dst < n) {
				int src_val = x[src];
				x[src] = x[dst];
				x[dst] += src_val;
			}
			__syncthreads();
		}
	}

	/* in-place scan over device array, doesn't start GPU Timer and assumes input is power of 2 */
	void scan_dev(int N, cu::cVec<int> *dev_data) {
		int log2n = ilog2ceil(N);
		int fullBlocksPerGrid = (N + blockSize - 1)/blockSize;
		for (int d = 0; d < log2n; d++) {
			int count = N / (1 << d);
			dim3 fullBlocksPerGrid((count + blockSize - 1) / blockSize);
			kern_up_sweep<<<fullBlocksPerGrid, blockSize>>>(d, N, dev_data->raw_ptr());
		}
		cu::set(dev_data->ptr() + N-1, 0, 1);
		for (int d = log2n - 1; d >= 0; d--) {
			int count = N / (1 << d);
			dim3 fullBlocksPerGrid((count + blockSize - 1) / blockSize);
			kern_down_sweep<<<fullBlocksPerGrid, blockSize>>>(d, N, dev_data->raw_ptr());
		}
	}

	void scan_dev2(int N, cu::cVec<int>* dev_data) {
		int fullBlocksPerGrid = (N/2 + blockSize - 1) / blockSize;
		kern_up_sweep2<<<fullBlocksPerGrid, blockSize>>>(N, dev_data->raw_ptr());
		cu::set(dev_data->ptr() + N-1, 0, 1);
		kern_down_sweep2<<<fullBlocksPerGrid, blockSize>>>(N, dev_data->raw_ptr());
	}


	/**
	 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
	 */
	void scan(int n, int *odata, const int *idata) {
		int log2n = ilog2ceil(n);
		int N = 1 << log2n;

		cu::cVec<int> dev_data(n, idata, N);

		timer().startGpuTimer();

		scan_dev2(N, &dev_data);

		timer().endGpuTimer();

		cu::copy(odata, dev_data.ptr(), n);
	}

	/**
	 * Performs stream compaction on idata, storing the result into odata.
	 * All zeroes are discarded.
	 *
	 * @param n      The number of elements in idata.
	 * @param odata  The array into which to store elements.
	 * @param idata  The array of elements to compact.
	 * @returns	The number of elements remaining after compaction.
	 */
	int compact(int n, int *odata, const int *idata) {
		int log2n = ilog2ceil(n);
		int N = 1 << log2n;

		cu::cVec<int> dev_idata(n, idata), dev_bdata(n), dev_sdata(N), dev_odata(n);

		timer().startGpuTimer();
		
		dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
		Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(n, dev_bdata.raw_ptr(), dev_idata.raw_ptr());

		cu::copy(dev_sdata.ptr(), dev_bdata.ptr(), n);
		scan_dev(N, &dev_sdata);

		Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(n, dev_odata.raw_ptr(), dev_idata.raw_ptr(), dev_bdata.raw_ptr(), dev_sdata.raw_ptr());

		timer().endGpuTimer();

		cu::copy(odata, dev_odata.ptr(), n);

		for (int i = 0; i < n; i++)
			if (!odata[i])
				return i;
		
		return n;
	}
}
}

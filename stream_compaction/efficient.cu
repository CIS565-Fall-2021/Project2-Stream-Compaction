#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction
{
    namespace Efficient
    {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer &timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernScanEfficientUpSweep(int n, int layer, int shift, int *data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n)
            {
                return;
            }
            if (index % (1 << (layer + 1)) == 0)
            {
                data[index + (1 << (layer + 1)) - 1] += data[index + (1 << layer) - 1];
            }
        }

        __global__ void kernScanEfficientDownSweep(int n, int layer, int max, int shift, int *data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n)
            {
                return;
            }
            if (index % (1 << (layer + 1)) == 0)
            {
                int t = data[index + (1 << layer) - 1];
                data[index + (1 << layer) - 1] = data[index + (1 << (layer + 1)) - 1];
                data[index + (1 << (layer + 1)) - 1] += t;
            }
        }

        __global__ void kernSetLastToZero(int n, int *data)
        {
            data[n - 1] = 0;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata)
        {
            //next power of 2
            int power = ilog2ceil(n);
            int size = 1 << power;
            int offset = size - n;
            dim3 fullBlocksPerGrid((size + blockSize - 1) / blockSize);
            int *buf;
            hipMalloc((void **)&buf, size * sizeof(int));
            hipMemset(buf, 0, size * sizeof(int));
            hipMemcpy(buf + offset, idata, n * sizeof(int), hipMemcpyHostToDevice);
            timer().startGpuTimer();
            // TODO
            for (int layer = 0; layer < power; layer++)
            {
                // invoke kernel
                int shift = 1 << layer;
                kernScanEfficientUpSweep<<<fullBlocksPerGrid, blockSize>>>(size, layer, shift, buf);
                hipDeviceSynchronize();
            }
            kernSetLastToZero<<<1, 1>>>(size, buf);
            hipDeviceSynchronize();
            for (int layer = power - 1; layer >= 0; layer--)
            {
                // invoke kernel
                int shift = 1 << layer;
                kernScanEfficientDownSweep<<<fullBlocksPerGrid, blockSize>>>(size, layer, power - 1, shift, buf);
                hipDeviceSynchronize();
                // checkCUDAErrorWithLine("hipDeviceSynchronize buf failed!");
            }
            timer().endGpuTimer();
            hipMemcpy(odata, buf + offset, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(buf);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata)
        {
            //next power of 2
            int power = ilog2ceil(n);
            int size = 1 << power;
            int offset = size - n;
            dim3 fullBlocksPerGrid((size + blockSize - 1) / blockSize);

            int *buf; // power of 2 0 padded copy of idata
            hipMalloc((void **)&buf, size * sizeof(int));
            hipMemset(buf, 0, size * sizeof(int));
            hipMemcpy(buf + offset, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int *bools, *indices, *tmpOut; //
            hipMalloc((void **)&bools, size * sizeof(int));
            hipMalloc((void **)&indices, size * sizeof(int));
            hipMalloc((void **)&tmpOut, size * sizeof(int));
            timer().startGpuTimer();
            // TODO
            // Map
            Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(size, bools, buf);
            Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(size, indices, buf);
            hipDeviceSynchronize();
            // Scan
            for (int layer = 0; layer < power; layer++)
            {
                // invoke kernel
                int shift = 1 << layer;
                kernScanEfficientUpSweep<<<fullBlocksPerGrid, blockSize>>>(size, layer, shift, indices);
                hipDeviceSynchronize();
            }
            kernSetLastToZero<<<1, 1>>>(size, indices);
            hipDeviceSynchronize();
            for (int layer = power - 1; layer >= 0; layer--)
            {
                // invoke kernel
                int shift = 1 << layer;
                kernScanEfficientDownSweep<<<fullBlocksPerGrid, blockSize>>>(size, layer, power - 1, shift, indices);
                hipDeviceSynchronize();
            }
            // Scatter
            Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(size, tmpOut, buf, bools, indices);
            hipDeviceSynchronize();
            timer().endGpuTimer();

            hipMemcpy(odata, tmpOut, n * sizeof(int), hipMemcpyDeviceToHost);
            int retSize;
            hipMemcpy(&retSize, indices + size - 1, sizeof(int), hipMemcpyDeviceToHost);
            int tmpLast;
            hipMemcpy(&tmpLast, buf + size - 1, sizeof(int), hipMemcpyDeviceToHost);
            retSize += (tmpLast != 0);

            hipFree(buf);
            hipFree(bools);
            hipFree(indices);
            hipFree(tmpOut);
            return retSize;
        }
    }
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int N, int offset, int *buffer){
            // offset: current depth of the tree
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
	        if (index >= (N >> offset)) return;
            int k = index << (offset);
	        buffer[k + (1 << (offset)) - 1] += buffer[k + (1 << (offset-1)) - 1];
        }

        __global__ void kernDownSweep(int N, int offset, int *buffer){
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= (N >> offset)) return;
            int k = index << offset;
            int tmp = buffer[k + (1 << offset) - 1];
            buffer[k + (1 << offset) - 1] += buffer[k + (1 << (offset - 1)) - 1];
            buffer[k + (1 << (offset - 1)) - 1] = tmp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int *buffer, fullBlocksPerGrid;
            // padded to the power of 2s and get the max depth D of the balanced tree
            int D = ilog2ceil(n);
            int N = 1 << D;
            hipMalloc((void**)&buffer, N * sizeof(int));
	        hipMemcpy(buffer, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            timer().startGpuTimer();
        
            for (int d= 0; d < D; d++) {
                fullBlocksPerGrid = ((N >> d) + blockSize - 1) / blockSize;
                kernUpSweep << <fullBlocksPerGrid, blockSize >> >(N, d+1, buffer);
            }
            timer().endGpuTimer();
            time += timer().getGpuElapsedTimeForPreviousOperation();
            hipMemset(buffer + N - 1, 0, sizeof(int));

            timer().startGpuTimer();
            for (int d = D-1; d >= 0; d--) {
                fullBlocksPerGrid = ((N >> (d+1)) + blockSize - 1) / blockSize;
                kernDownSweep << <fullBlocksPerGrid, blockSize >> >(N, d+1, buffer);
            }    
            timer().endGpuTimer();
            time += timer().getGpuElapsedTimeForPreviousOperation();
        }   
        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // Work-Efficient Compact
            time = 0;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            // init 
            int *bools, *indices, *in, *out;
            // memory allocation 
            hipMalloc((void**)&bools, n * sizeof(int));
            hipMalloc((void**)&indices, n * sizeof(int));
            hipMalloc((void**)&in, n * sizeof(int));
            hipMalloc((void**)&out, n * sizeof(int));
            // copy to device
            hipMemcpy(in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, bools, in);
            timer().endGpuTimer();
            time += timer().getGpuElapsedTimeForPreviousOperation();
            // copy to host
            hipMemcpy(odata, bools, n * sizeof(int), hipMemcpyDeviceToHost);
            // work efficient scan
            scan(n, odata, odata);
            int lenCompacted = odata[n - 1];
            // copy to device
            hipMemcpy(indices, odata, n * sizeof(int), hipMemcpyHostToDevice);
            timer().startGpuTimer();
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, out, in, bools, indices);
            timer().endGpuTimer();
            time += timer().getGpuElapsedTimeForPreviousOperation();
            printf("Work-Efficient scan: %f ms\n", time);
            hipMemcpy(odata, out, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(bools);
            hipFree(indices);
            hipFree(in);
            hipFree(out);
            
            return lenCompacted;
        }
    }
}

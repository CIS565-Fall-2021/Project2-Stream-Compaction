#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {
using StreamCompaction::Common::PerformanceTimer;
PerformanceTimer &timer() {
  static PerformanceTimer timer;
  return timer;
}

__global__ void kernScanInclusive(int n, int *odata, int *idata) {
  int tid   = threadIdx.x;
  int bdim  = blockDim.x;
  int id    = blockIdx.x * bdim + tid;
  int log2n = ilog2ceil((n < bdim) ? n : bdim);
  if (id < n) {
    // upsweep
    for (int d = 0; d < log2n; ++d) {
      if (id % (1 << (d + 1)) == 0) {
        idata[id + (1 << (d + 1)) - 1] += idata[id + (1 << d) - 1];
      }
      __syncthreads();
    }

    // last thread remembers and sets reduction sum after downsweep
    int reduction_sum = 0;
    if (tid == bdim - 1 || id == n - 1) {
      reduction_sum = idata[id];
      idata[id]     = 0;
    }
    __syncthreads();

    // downsweep
    for (int d = log2n - 1; d >= 0; --d) {
      if (id % (1 << (d + 1)) == 0) {
        int temp                 = idata[id + (1 << d) - 1];
        idata[id + (1 << d) - 1] = idata[id + (1 << (d + 1)) - 1];
        idata[id + (1 << (d + 1)) - 1] += temp;
      }
      __syncthreads();
    }

    // turn exclusive scan into inclusive scan
    if (tid == bdim - 1 || id == n - 1) {
      odata[id] = reduction_sum;
    } else {
      odata[id] = idata[id + 1];
    }
  }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
  const unsigned int grid_size =
      (n + Common::block_size - 1) / Common::block_size;
  const int n_pad = 1 << (ilog2ceil(n));

  // allocate input/output device data
  int *dev_idata, *dev_odata;
  hipMalloc((void **)&dev_idata, n_pad * sizeof(int));
  hipMalloc((void **)&dev_odata, n_pad * sizeof(int));
  checkCUDAError("hipMalloc dev_idata, dev_odata failed!");

  hipMemset(dev_idata, 0, n_pad * sizeof(int));
  checkCUDAError("hipMemset dev_idata to 0 failed!");
  hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy dev_idata from idata failed!");

  // allocate block reduction device data
  int *dev_block_offset_inclusive, *dev_block_offset_exclusive;
  hipMalloc((void **)&dev_block_offset_inclusive, grid_size * sizeof(int));
  hipMalloc((void **)&dev_block_offset_exclusive, grid_size * sizeof(int));

  /******* KERNEL INVOCATION *******/
  dim3 dimGrid{grid_size}, dimBlock{Common::block_size};
  timer().startGpuTimer();
  kernScanInclusive<<<dimGrid, dimBlock>>>(n_pad, dev_odata, dev_idata);
  Common::kernExtractLastElementPerBlock<<<dimGrid, dimBlock>>>(
      n_pad, dev_block_offset_exclusive, dev_odata);
  kernScanInclusive<<<1, dimBlock>>>(grid_size, dev_block_offset_inclusive,
                                     dev_block_offset_exclusive);
  Common::kernShiftToExclusive<<<1, dimBlock>>>(
      grid_size, dev_block_offset_exclusive, dev_block_offset_inclusive);
  Common::kernAddOffsetPerBlock<<<dimGrid, dimBlock>>>(
      n_pad, dev_idata, dev_block_offset_exclusive, dev_odata);
  Common::kernShiftToExclusive<<<dimGrid, dimBlock>>>(n_pad, dev_odata,
                                                      dev_idata);
  timer().endGpuTimer();
  /*********************************/

  hipDeviceSynchronize();
  hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
  checkCUDAError("hipMemcpy odata from dev_idata failed!");
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
  timer().startGpuTimer();
  // TODO
  timer().endGpuTimer();
  return -1;
}
}  // namespace Efficient
}  // namespace StreamCompaction

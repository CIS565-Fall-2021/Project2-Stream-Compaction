#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {
using StreamCompaction::Common::PerformanceTimer;
PerformanceTimer &timer() {
  static PerformanceTimer timer;
  return timer;
}

__global__ void kernScanInclusive(int n, int *odata, int *idata) {
  int tid   = threadIdx.x;
  int bdim  = blockDim.x;
  int id    = blockIdx.x * bdim + tid;
  int log2n = ilog2ceil((n < bdim) ? n : bdim);
  if (id < n) {
    // upsweep
    for (int d = 0; d < log2n; ++d) {
      if (id % (1 << (d + 1)) == 0) {
        idata[id + (1 << (d + 1)) - 1] += idata[id + (1 << d) - 1];
      }
      __syncthreads();
    }

    // last thread remembers and sets reduction sum after downsweep
    int reduction_sum = 0;
    if (tid == bdim - 1 || id == n - 1) {
      reduction_sum = idata[id];
      idata[id]     = 0;
    }
    __syncthreads();

    // downsweep
    for (int d = log2n - 1; d >= 0; --d) {
      if (id % (1 << (d + 1)) == 0) {
        int temp                 = idata[id + (1 << d) - 1];
        idata[id + (1 << d) - 1] = idata[id + (1 << (d + 1)) - 1];
        idata[id + (1 << (d + 1)) - 1] += temp;
      }
      __syncthreads();
    }

    // turn exclusive scan into inclusive scan
    if (tid == bdim - 1 || id == n - 1) {
      odata[id] = reduction_sum;
    } else {
      odata[id] = idata[id + 1];
    }
  }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
  const unsigned int grid_size =
      (n + Common::block_size - 1) / Common::block_size;
  const int n_pad = 1 << (ilog2ceil(n));

  // allocate input/output device data
  int *dev_idata, *dev_odata;
  hipMalloc((void **)&dev_idata, n_pad * sizeof(int));
  hipMalloc((void **)&dev_odata, n_pad * sizeof(int));
  checkCUDAError("hipMalloc dev_idata, dev_odata failed!");

  hipMemset(dev_idata, 0, n_pad * sizeof(int));
  checkCUDAError("hipMemset dev_idata to 0 failed!");
  hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy dev_idata from idata failed!");

  // allocate block reduction device data
  int *dev_block_offset_inclusive, *dev_block_offset_exclusive;
  hipMalloc((void **)&dev_block_offset_inclusive, grid_size * sizeof(int));
  hipMalloc((void **)&dev_block_offset_exclusive, grid_size * sizeof(int));

  /******* KERNEL INVOCATION *******/
  dim3 dimGrid{grid_size}, dimBlock{Common::block_size};
  timer().startGpuTimer();
  kernScanInclusive<<<dimGrid, dimBlock>>>(n_pad, dev_odata, dev_idata);
  Common::kernExtractLastElementPerBlock<<<dimGrid, dimBlock>>>(
      n_pad, dev_block_offset_exclusive, dev_odata);
  kernScanInclusive<<<1, dimBlock>>>(grid_size, dev_block_offset_inclusive,
                                     dev_block_offset_exclusive);
  Common::kernShiftToExclusive<<<1, dimBlock>>>(
      grid_size, dev_block_offset_exclusive, dev_block_offset_inclusive);
  Common::kernAddOffsetPerBlock<<<dimGrid, dimBlock>>>(
      n_pad, dev_idata, dev_block_offset_exclusive, dev_odata);
  Common::kernShiftToExclusive<<<dimGrid, dimBlock>>>(n_pad, dev_odata,
                                                      dev_idata);
  timer().endGpuTimer();
  /*********************************/

  hipDeviceSynchronize();
  hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
  checkCUDAError("hipMemcpy odata from dev_idata failed!");
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
  const unsigned int grid_size =
      (n + Common::block_size - 1) / Common::block_size;
  const int n_pad = 1 << (ilog2ceil(n));

  // Device data allocation
  int *dev_idata, *dev_odata;
  hipMalloc((void **)&dev_idata, n_pad * sizeof(int));
  hipMalloc((void **)&dev_odata, n_pad * sizeof(int));
  checkCUDAError("hipMalloc dev_idata, dev_odata failed!");
  hipMemset(dev_idata, 0, n_pad * sizeof(int));
  checkCUDAError("hipMemset dev_idata to 0 failed!");

  hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy dev_idata from idata failed!");

  // helper data allocation
  int *dev_indices, *dev_bools, *dev_buffer;
  hipMalloc((void **)&dev_indices, n_pad * sizeof(int));
  hipMalloc((void **)&dev_bools, n_pad * sizeof(int));
  hipMalloc((void **)&dev_buffer, n_pad * sizeof(int));
  checkCUDAError("hipMalloc dev_indices, dev_bools, dev_buffer failed!");

  int *dev_offset_inclusive, *dev_offset_exclusive;
  hipMalloc((void **)&dev_offset_exclusive, grid_size * sizeof(int));
  hipMalloc((void **)&dev_offset_inclusive, grid_size * sizeof(int));
  checkCUDAError(
      "hipMalloc dev_offset_inclusive, dev_offset_exclusive failed!");

  /******* KERNEL INVOCATION *******/
  dim3 dimGrid{grid_size}, dimBlock{Common::block_size};
  timer().startGpuTimer();
  Common::kernMapToBoolean<<<dimGrid, dimBlock>>>(n_pad, dev_bools, dev_idata);
  hipDeviceSynchronize();
  hipMemcpy(dev_buffer, dev_bools, n_pad * sizeof(int),
             hipMemcpyDeviceToDevice);
  kernScanInclusive<<<dimGrid, dimBlock>>>(n_pad, dev_odata, dev_buffer);
  Common::kernExtractLastElementPerBlock<<<dimGrid, dimBlock>>>(
      n_pad, dev_offset_exclusive, dev_odata);
  kernScanInclusive<<<1, dimBlock>>>(grid_size, dev_offset_inclusive,
                                     dev_offset_exclusive);
  Common::kernShiftToExclusive<<<1, dimBlock>>>(grid_size, dev_offset_exclusive,
                                                dev_offset_inclusive);
  Common::kernAddOffsetPerBlock<<<dimGrid, dimBlock>>>(
      n_pad, dev_buffer, dev_offset_exclusive, dev_odata);
  Common::kernShiftToExclusive<<<dimGrid, dimBlock>>>(n_pad, dev_indices,
                                                      dev_buffer);
  Common::kernScatter<<<dimGrid, dimBlock>>>(n_pad, dev_odata, dev_idata,
                                             dev_bools, dev_indices);
  timer().endGpuTimer();
  /*********************************/

  // transfer output data to CPU & analyze
  hipDeviceSynchronize();
  hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
  checkCUDAError("hipMemcpy odata from dev_odata failed!");

  // calculate num. of elements after compaction
  int *indices = (int *)malloc(n * sizeof(int));
  int *bools   = (int *)malloc(n * sizeof(int));
  hipMemcpy(indices, dev_indices, n * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(bools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);
  checkCUDAError(
      "hipMemcpy indices from dev_indices, bools from dev_bools failed!");
  int compact_len = indices[n - 1] + bools[n - 1];
  free(indices);
  free(bools);

  return compact_len;
}
}  // namespace Efficient
}  // namespace StreamCompaction

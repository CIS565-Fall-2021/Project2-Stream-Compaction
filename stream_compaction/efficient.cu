#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

// Block size used for CUDA kernel launch
#define BLOCK_SIZE 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // Add each value at (index+2^(d+1)-1) to the value at (index+2^d-1) in place
        __global__ void kern_reduction(int n, int d, int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            // Only for multiple of 2^(d+1)
            if ((index & ((1 << (d + 1)) - 1)) == 0) {
                idata[index + (1 << (d + 1)) - 1] += idata[index + (1 << d) - 1];
            }
        }

        // Up-Sweep phase of efficient scan
        void up_sweep(int n, int* idata) {
            dim3 fullBlocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

            // Reduction for log(n) times
            for (int d = 0; d < ilog2ceil(n); d++) {
                kern_reduction << <fullBlocksPerGrid, BLOCK_SIZE >> > (n, d, idata);
                checkCUDAError("kern_reduction failed!");
            }
        }

        // Swap values of left and right children, then add value of left to right
        __global__ void kern_child_swap_add(int n, int d, int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            // Only for multiple of 2^(d+1)
            if ((index & ((1 << (d + 1)) - 1)) == 0) {
                int temp = idata[index + (1 << d) - 1];
                idata[index + (1 << d) - 1] = idata[index + (1 << (d + 1)) - 1];
                idata[index + (1 << (d + 1)) - 1] += temp;
            }
        }

        // Set last element to zero
        __global__ void kern_clear_root(int n, int *idata) {
            idata[n - 1] = 0;
        }

        // Down-Sweep phase of efficient scan
        void down_sweep(int n, int* idata) {
            dim3 fullBlocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

            // Set root to zero
            kern_clear_root << <1, 1 >> > (n, idata);
            checkCUDAError("kern_clear_root failed!");

            // log(n) passes
            for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
                kern_child_swap_add << <fullBlocksPerGrid, BLOCK_SIZE >> > (n, d, idata);
                checkCUDAError("kern_child_swap_add failed!");
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool timing_on) {
            // Create device array
            // Rounded to next power of two
            int round_n = 1 << ilog2ceil(n);
            int *dev_array;
            hipMalloc((void**)&dev_array, round_n * sizeof(int));
            checkCUDAError("hipMalloc failed!");

            // Copy data to GPU
            hipMemcpy(dev_array, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy failed!");

            if (timing_on) {
                timer().startGpuTimer();
            }

            up_sweep(round_n, dev_array);

            down_sweep(round_n, dev_array);

            if (timing_on) {
                timer().endGpuTimer();
            }

            // Copy data back
            hipMemcpy(odata, dev_array, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy back failed!");

            // Cleanup
            hipFree(dev_array);
            checkCUDAError("hipFree failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            dim3 fullBlocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

            // Used for computing the number of elements remaining after compaction
            int *last_elements = new int[2];

            // Create device array
            int *dev_array;
            int *dev_bool_buffer;
            int *dev_scan_buffer;
            int *dev_res;
            hipMalloc((void **)&dev_array, n * sizeof(int));
            hipMalloc((void **)&dev_bool_buffer, n * sizeof(int));
            hipMalloc((void **)&dev_scan_buffer, n * sizeof(int));
            hipMalloc((void **)&dev_res, n * sizeof(int));
            checkCUDAError("hipMalloc failed!");

            // Copy data to GPU
            hipMemcpy(dev_array, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy failed!");

            timer().startGpuTimer();

            // Set 1 for non-zero elements
            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, BLOCK_SIZE >> > (n, dev_bool_buffer, dev_array);
            checkCUDAError("kernMapToBoolean failed!");

            // Scan
            scan(n, dev_scan_buffer, dev_bool_buffer, false);

            // Scatter
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, BLOCK_SIZE >> > (n, dev_res, dev_array, dev_bool_buffer, dev_scan_buffer);
            checkCUDAError("kernScatter failed!");

            timer().endGpuTimer();

            // Fetch last element of bool array and scan array respectively
            hipMemcpy(last_elements, dev_bool_buffer + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(last_elements + 1, dev_scan_buffer + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy back failed!");

            // Compute the number of elements remaining after compaction
            int num_element = last_elements[0] + last_elements[1];
            free(last_elements);

            // Copy data back
            hipMemcpy(odata, dev_res, sizeof(int) * num_element, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy back failed!");

            // Cleanup
            hipFree(dev_array);
            hipFree(dev_bool_buffer);
            hipFree(dev_scan_buffer);
            hipFree(dev_res);
            checkCUDAError("hipFree failed!");

            return num_element;
        }
    }



    namespace Efficient_Upgraded {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer &timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // Add each value at (index+2^(d+1)-1) to the value at (index+2^d-1) in place
        __global__ void kern_reduction(int n, int d, int *idata) {
            unsigned long int index = threadIdx.x + (blockIdx.x * blockDim.x);

            // Index hack
            // Make use of all threads
            index *= (1 << (d + 1));

            if (index >= n) {
                return;
            }

            // 'index' is now multiple of 2^(d+1)
            idata[index + (1 << (d + 1)) - 1] += idata[index + (1 << d) - 1];
        }

        // Up-Sweep phase of efficient scan
        void up_sweep(int n, int *idata) {
            // Number of active elements in array
            int act_n = n;

            // Reduction for log(n) times
            for (int d = 0; d < ilog2ceil(n); d++) {
                // Halve the number of blocks launched in each turn
                act_n /= 2;
                dim3 fullBlocksPerGrid((act_n + BLOCK_SIZE - 1) / BLOCK_SIZE);
                kern_reduction << <fullBlocksPerGrid, BLOCK_SIZE >> > (n, d, idata);
                checkCUDAError("kern_reduction failed!");
            }
        }

        // Swap values of left and right children, then add value of left to right
        __global__ void kern_child_swap_add(int n, int d, int *idata) {
            unsigned long int index = threadIdx.x + (blockIdx.x * blockDim.x);

            // Index hack
            // Make use of all threads
            index *= (1 << (d + 1));

            if (index >= n) {
                return;
            }

            // 'index' is now multiple of 2^(d+1)
            int temp = idata[index + (1 << d) - 1];
            idata[index + (1 << d) - 1] = idata[index + (1 << (d + 1)) - 1];
            idata[index + (1 << (d + 1)) - 1] += temp;
        }

        // Set last element to zero
        __global__ void kern_clear_root(int n, int *idata) {
            idata[n - 1] = 0;
        }

        // Down-Sweep phase of efficient scan
        void down_sweep(int n, int *idata) {
            // Set root to zero
            kern_clear_root << <1, 1 >> > (n, idata);
            checkCUDAError("kern_clear_root failed!");

            // Number of active elements in array
            int act_n = n / (1 << (ilog2ceil(n) + 1)) < 1 ? 1 : n / (1 << (ilog2ceil(n) + 1));

            // log(n) passes
            for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
                // Double the number of blocks launched in each turn
                act_n *= 2;
                dim3 fullBlocksPerGrid((act_n + BLOCK_SIZE - 1) / BLOCK_SIZE);
                kern_child_swap_add << <fullBlocksPerGrid, BLOCK_SIZE >> > (n, d, idata);
                checkCUDAError("kern_child_swap_add failed!");
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool timing_on) {
            // Create device array
            // Rounded to next power of two
            int round_n = 1 << ilog2ceil(n);
            int *dev_array;
            hipMalloc((void **)&dev_array, round_n * sizeof(int));
            checkCUDAError("hipMalloc failed!");

            // Copy data to GPU
            hipMemcpy(dev_array, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy failed!");

            if (timing_on) {
                timer().startGpuTimer();
            }

            up_sweep(round_n, dev_array);

            down_sweep(round_n, dev_array);

            if (timing_on) {
                timer().endGpuTimer();
            }

            // Copy data back
            hipMemcpy(odata, dev_array, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy back failed!");

            // Cleanup
            hipFree(dev_array);
            checkCUDAError("hipFree failed!");
        }
    }



    namespace Efficient_Shared {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer &timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // Perform inclusive scan in place on arr, and store sum of this block to sum
        // Use shared memory to reduce memory access latency
        // Notice that this can only process within ONE block, so n is at most as TWICE as max number of threads in a block
        // 
        // Reference: GPU Gem Ch 39 Example 39.2
        // https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
        __global__ void kern_prescan_inclusive(int n, int *arr, int *sum) {
            extern __shared__ int shared_buffer[];

            int index = threadIdx.x;
            int act_index = threadIdx.x + blockIdx.x * blockDim.x;
            int block_index = blockIdx.x;
            int offset = 1;

            // Copy data to shared memory
            shared_buffer[index * 2] = arr[act_index * 2];
            shared_buffer[index * 2 + 1] = arr[act_index * 2 + 1];

            // Up-Sweep
            for (int d = n >> 1; d > 0; d >>= 1) {
                // Synchronize all threads at each turn
                __syncthreads();

                // Reduction
                if (index < d) {
                    shared_buffer[offset * (2 * index + 2) - 1] += shared_buffer[offset * (2 * index + 1) - 1];
                }

                // At next turn, double the stride to access
                offset *= 2;
            }

            // Clear root
            if (index == 0) {
                shared_buffer[n - 1] = 0;
            }

            // Down-Sweep
            for (int d = 1; d < n; d *= 2) {
                // At next turn, halve the stride to access
                offset >>= 1;

                // Synchronize all threads at each turn
                __syncthreads();

                // Swap values of left and right children, then add value of left to right
                if (index < d) {
                    int temp = shared_buffer[offset * (2 * index + 1) - 1];
                    shared_buffer[offset * (2 * index + 1) - 1] = shared_buffer[offset * (2 * index + 2) - 1];
                    shared_buffer[offset * (2 * index + 2) - 1] += temp;
                }
            }

            __syncthreads();

            // Copy data back
            arr[act_index * 2] = shared_buffer[index * 2 + 1];

            // Write sum of block
            if (index * 2 + 2 == n) {
                arr[act_index * 2 + 1] += shared_buffer[index * 2 + 1];
                sum[block_index] = arr[act_index * 2 + 1];
            }
            else {
                arr[act_index * 2 + 1] = shared_buffer[index * 2 + 2];
            }
        }

        // Perform exclusive scan in place on arr
        // Use shared memory to reduce memory access latency
        // Notice that this can only process within ONE block, so n is at most as TWICE as max number of threads in a block
        // 
        // Reference: GPU Gem Ch 39 Example 39.2
        // https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
        __global__ void kern_prescan_exclusive(int n, int *arr) {
            extern __shared__ int shared_buffer[];

            int index = threadIdx.x;
            int offset = 1;

            // Copy data to shared memory
            shared_buffer[index * 2] = arr[index * 2];
            shared_buffer[index * 2 + 1] = arr[index * 2 + 1];

            // Up-Sweep
            for (int d = n >> 1; d > 0; d >>= 1) {
                // Synchronize all threads at each turn
                __syncthreads();

                // Reduction
                if (index < d) {
                    shared_buffer[offset * (2 * index + 2) - 1] += shared_buffer[offset * (2 * index + 1) - 1];
                }

                // At next turn, double the stride to access
                offset *= 2;
            }

            // Clear root
            if (index == 0) {
                shared_buffer[n - 1] = 0;
            }

            // Down-Sweep
            for (int d = 1; d < n; d *= 2) {
                // At next turn, halve the stride to access
                offset >>= 1;

                // Synchronize all threads at each turn
                __syncthreads();

                // Swap values of left and right children, then add value of left to right
                if (index < d) {
                    int temp = shared_buffer[offset * (2 * index + 1) - 1];
                    shared_buffer[offset * (2 * index + 1) - 1] = shared_buffer[offset * (2 * index + 2) - 1];
                    shared_buffer[offset * (2 * index + 2) - 1] += temp;
                }
            }

            __syncthreads();

            // Copy data back
            arr[index * 2] = shared_buffer[index * 2];
            arr[index * 2 + 1] = shared_buffer[index * 2 + 1];
        }

        // Add block increments to each element in the corresponding block
        __global__ void kern_add_increment(int n, int *arr, int *sum) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int block_index = blockIdx.x;
            if (index >= n) {
                return;
            }

            arr[index * 2] += sum[block_index];
            arr[index * 2 + 1] += sum[block_index];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool timing_on) {
            int num_blocks = (n + (2 * BLOCK_SIZE) - 1) / (2 * BLOCK_SIZE);
            int round_num_blocks = 1 << ilog2ceil(num_blocks);

            // Create device array
            int *dev_array;
            int *dev_block_sums;
            hipMalloc((void **)&dev_array, n * sizeof(int));
            hipMalloc((void **)&dev_block_sums, round_num_blocks * sizeof(int));
            checkCUDAError("hipMalloc failed!");

            // Copy data to GPU
            hipMemcpy(dev_array, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy failed!");

            if (timing_on) {
                timer().startGpuTimer();
            }

            dim3 halfBlocksPerGrid(num_blocks);
            // Scan each block and record block sums
            kern_prescan_inclusive << <halfBlocksPerGrid, BLOCK_SIZE, (2 * BLOCK_SIZE) * sizeof(int) >> > ((2 * BLOCK_SIZE), dev_array, dev_block_sums);
            checkCUDAError("kern_prescan failed!");

            // Scan block sums
            Efficient_Upgraded::up_sweep(round_num_blocks, dev_block_sums);

            Efficient_Upgraded::down_sweep(round_num_blocks, dev_block_sums);

            // Add increments
            kern_add_increment << <halfBlocksPerGrid, BLOCK_SIZE >> > (n, dev_array, dev_block_sums);
            checkCUDAError("kern_add_increment failed!");

            // Set identity
            odata[0] = 0;

            if (timing_on) {
                timer().endGpuTimer();
            }

            // Copy data back
            // Shift inclusive scan to exclusive scan           
            hipMemcpy(odata + 1, dev_array, sizeof(int) * (n - 1), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy back failed!");

            // Cleanup
            hipFree(dev_array);
            hipFree(dev_block_sums);
            checkCUDAError("hipFree failed!");
        }
    }
}

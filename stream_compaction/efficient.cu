#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernWorkEfficientReduceStep(int n, int stride, int* data) {
          int index = threadIdx.x + (blockIdx.x * blockDim.x);
          if (index >= stride && index < n) {
            data[index] += data[index - stride];
          }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            /*
            dim3 fullBlocksPerGrid = ((n + blockSize - 1) / blockSize);

            int* dev_buf0;
            hipMalloc((void**)&dev_buf0, n * sizeof(int));
            checkCUDAError("hipMalloc dev_buf0 failed!");

            int* dev_buf1;
            hipMalloc((void**)&dev_buf1, n * sizeof(int));
            checkCUDAError("hipMalloc dev_buf0 failed!");

            hipMemcpy(dev_buf0, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to device failed!");

            for (int offset = 1; offset < n; offset <<= 1) {
              kernNaiveScanStep << <fullBlocksPerGrid, blockSize >> > (n, offset, dev_buf1, dev_buf0);
              checkCUDAError("kernNaiveScanStep failed!");

              std::swap(dev_buf0, dev_buf1);
            }

            hipMemcpy(&odata[1], dev_buf0, sizeof(int) * (n - 1), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from device failed!");

            std::vector<int> odataView(n);
            memcpy(odataView.data(), odata, sizeof(int) * n);

            std::vector<int> idataView(n);
            memcpy(idataView.data(), idata, sizeof(int) * n);

            hipFree(dev_buf0);
            checkCUDAError("hipFree dev_buf0 failed!");

            hipFree(dev_buf1);
            checkCUDAError("hipFree dev_buf1 failed!");
            */
            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}

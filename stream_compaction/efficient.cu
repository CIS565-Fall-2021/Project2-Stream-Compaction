#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        
        __global__ void kernWorkEfficientUpSweepStep(int n, int stride, int* data) {
          int index = 2 * stride * (threadIdx.x + (blockIdx.x * blockDim.x)) - 1;
          if (index >= stride && index < n) {
            data[index] += data[index - stride];
          }
        }
        
        __global__ void kernWorkEfficientDownSweepStep(int n, int stride, int* data) {
          int index = 2 * stride * (threadIdx.x + (blockIdx.x * blockDim.x)) - 1;
          if (index >= stride && index < n) {
            int oldValue = data[index];
            data[index] += data[index - stride];
            data[index - stride] = oldValue;
          }
        }

        /**
         * Performs prefix-sum (aka scan) on the buffer in place. Expects a padding to keep the length a power of 2.
         */
        void _scan(int n, int *dev_buf) {
            dim3 fullBlocksPerGrid = ((n + blockSize - 1) / blockSize);

            // up-sweep phase
            for (int stride = 1; stride < n; stride <<= 1) {
              kernWorkEfficientUpSweepStep << <fullBlocksPerGrid, blockSize >> > (n, stride, dev_buf);
              checkCUDAError("kernWorkEfficientUpSweepStep failed!");
            }

            // down-sweep phase
            hipMemset(&dev_buf[n - 1], 0, sizeof(int));
            for (int stride = n >> 1; stride > 0; stride >>= 1) {
              kernWorkEfficientDownSweepStep << <fullBlocksPerGrid, blockSize >> > (n, stride, dev_buf);
              checkCUDAError("kernWorkEfficientDownSweepStep failed!");
            }
        }
         
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
          int nPow2 = 1 << ilog2ceil(n);

          int* dev_buf;
          hipMalloc((void**)&dev_buf, sizeof(int) * nPow2);
          checkCUDAError("hipMalloc dev_buf failed!");

          hipMemcpy(dev_buf, idata, sizeof(int) * n, hipMemcpyHostToDevice);
          checkCUDAError("hipMemcpy to device failed!");

          timer().startGpuTimer();

          _scan(nPow2, dev_buf);

          timer().endGpuTimer();

          hipMemcpy(odata, dev_buf, sizeof(int) * n, hipMemcpyDeviceToHost);
          checkCUDAError("hipMemcpy from device failed!");

          hipFree(dev_buf);
          checkCUDAError("hipFree dev_buf failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            dim3 fullBlocksPerGrid = ((n + blockSize - 1) / blockSize);

            int nPow2 = 1 << ilog2ceil(n);

            int* dev_input;
            hipMalloc((void**)&dev_input, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_input failed!");

            hipMemcpy(dev_input, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy input to device failed!");

            int* dev_bools;
            hipMalloc((void**)&dev_bools, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_bools failed!");

            int* dev_indices;
            hipMalloc((void**)&dev_indices, sizeof(int) * nPow2);
            checkCUDAError("hipMalloc dev_indices failed!");

            timer().startGpuTimer();

            Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bools, dev_input);
            checkCUDAError("kernMapToBoolean failed!");

            hipMemcpy(dev_indices, dev_bools, sizeof(int) * n, hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy from device to device failed!");

            _scan(nPow2, dev_indices);

            int count = 0;
            hipMemcpy(&count, &dev_indices[n - 1], sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from device failed!");
            count += idata[n - 1] != 0;

            int* dev_output;
            hipMalloc((void**)&dev_output, sizeof(int) * count);
            checkCUDAError("hipMalloc dev_output failed!");

            Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_output, dev_input, dev_bools, dev_indices);
            checkCUDAError("kernScatter failed!");

            timer().endGpuTimer();

            hipMemcpy(odata, dev_output, sizeof(int) * count, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy output to host failed!");

            hipFree(dev_input);
            checkCUDAError("hipFree dev_input failed!");

            hipFree(dev_output);
            checkCUDAError("hipFree dev_output failed!");

            hipFree(dev_bools);
            checkCUDAError("hipFree dev_bools failed!");

            hipFree(dev_indices);
            checkCUDAError("hipFree dev_indices failed!");

            return count;
        }
    }
}

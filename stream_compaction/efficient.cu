#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#ifndef BLOCKSIZE
#define BLOCKSIZE 128
#endif // !BLOCKSIZE

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int *odata, int d) {

            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index > n) {
                return;
            }

            int k = index * (1 << (d + 1));

			odata[k + ((1<<(d+1))-1)] = odata[k + (1 << d) - 1] + odata[k + (1 << (d+1)) - 1];
            __syncthreads();
        }

        __global__ void kernDownSweep(int n, int *odata, int d) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index > n) {
                return;
            }

            int k = index * (1 << (d + 1));
            int t = odata[k + (1 << d) - 1];
            odata[k + (1 << d) - 1] = odata[k + (1 << (d + 1)) - 1];
            odata[k + (1 << (d + 1)) - 1] += t;
            __syncthreads();
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
           
            int* dev_readable; 
            int* dev_odata; 

            // pad to a power of 2
            int paddedN = 1 << ilog2ceil(n);

            hipMalloc((void**)&dev_odata, paddedN * sizeof(int));

            // write n items to the GPU array, the total length is `paddedN`, meaning arr[n:paddedN] are 0 
            hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            // --- up sweep ---
            for (int d = 0; d < log2(paddedN); d++) {
				int numThreads = ((paddedN - 1) / (1 << (d + 1))) + 1;
				kernUpSweep <<<numThreads, BLOCKSIZE>>> (numThreads, dev_odata, d);
				checkCUDAErrorFn("upsweep failed", "efficent.cu", 50);
				hipDeviceSynchronize();
				//hipMemcpy(odata, dev_odata, paddedN * sizeof(int), hipMemcpyDeviceToHost);
            }

            // --- down sweep ---
            // insert 0 at the end of the in-progress output
            int ZERO = 0;
            hipMemcpy(dev_odata + paddedN - 1, &ZERO, sizeof(int), hipMemcpyHostToDevice);
            for (int d = log2(paddedN - 1); d >= 0; d--) {
				int numThreads = ((paddedN - 1) / (1 << (d + 1))) + 1;
				kernDownSweep <<<numThreads, BLOCKSIZE>>> (numThreads, dev_odata, d);
				checkCUDAErrorFn("downsweep failed", "efficent.cu", 70);
				hipDeviceSynchronize();
				//hipMemcpy(odata, dev_odata, paddedN * sizeof(int), hipMemcpyDeviceToHost);
            }

            timer().endGpuTimer();

            // this is an exclusive scan, so the first elem should be 0
            // and we shift everything (except the last elem) one index right
            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_odata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int paddedN = 1<<ilog2ceil(n);
            int* dev_odata;
            int* dev_idata;
            int* dev_hasElem;
            int* dev_indices;

            hipMalloc((void**)&dev_odata, paddedN * sizeof(int));
            hipMalloc((void**)&dev_idata, paddedN * sizeof(int));
            hipMalloc((void**)&dev_hasElem, paddedN * sizeof(int));
            hipMalloc((void**)&dev_indices, paddedN * sizeof(int));
            
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            Common::kernMapToBoolean<<<n, BLOCKSIZE >>>(n, dev_hasElem, dev_idata);

            hipMemcpy(dev_indices, dev_hasElem, n * sizeof(int), hipMemcpyDeviceToDevice);

            // --- scan ---
            // scanning the `hasElem` "boolean" array yields an array of indices
            // idata[i] should be assigned to (if hasElem[i] is truthy).
            // why copy (most of the) code from scan() instead of calling it?
            // to avoid redundant hipMalloc/hipMemcpy calls

            // --- up sweep ---
            for (int d = 0; d < log2(paddedN); d++) {
				int numThreads = ((paddedN - 1) / (1 << (d + 1))) + 1;
				kernUpSweep <<<numThreads, BLOCKSIZE>>> (numThreads, dev_indices, d);
				checkCUDAErrorFn("upsweep failed", "efficent.cu", 50);
				hipDeviceSynchronize();
				//hipMemcpy(odata, dev_odata, paddedN * sizeof(int), hipMemcpyDeviceToHost);
            }

            // --- down sweep ---
            // insert 0 at the end of the in-progress output
            int ZERO = 0;
            hipMemcpy(dev_indices + paddedN - 1, &ZERO, sizeof(int), hipMemcpyHostToDevice);
            for (int d = log2(paddedN - 1); d >= 0; d--) {
				int numThreads = ((paddedN - 1) / (1 << (d + 1))) + 1;
				kernDownSweep <<<numThreads, BLOCKSIZE>>> (numThreads, dev_indices, d);
				checkCUDAErrorFn("downsweep failed", "efficent.cu", 70);
				hipDeviceSynchronize();
				//hipMemcpy(odata, dev_odata, paddedN * sizeof(int), hipMemcpyDeviceToHost);
            }

            // --- scatter ---
            // assign idata -> odata based on the indices calculated by the scan

            Common::kernScatter<<<n, BLOCKSIZE>>>(n, dev_odata, dev_idata, dev_hasElem, dev_indices);

            timer().endGpuTimer();

            // get the max index for return purposes. This will be whatever is at the end
            // of our scattered index array
            int maxIndex;
            hipMemcpy(&maxIndex, dev_indices + paddedN - 1, sizeof(int), hipMemcpyDeviceToHost);
            
            hipMemcpy(odata, dev_odata, maxIndex * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_odata);

            return maxIndex;
        }
    }
}

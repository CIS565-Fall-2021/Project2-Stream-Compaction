#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // Create thrust vector and cast to GPU
            thrust::host_vector<int> thrust_idata(idata, idata + n);
            thrust::device_vector<int> dev_thrust_idata = thrust_idata;
            thrust::device_vector<int> dev_thrust_odata(n);
            
            timer().startGpuTimer();

            thrust::exclusive_scan(dev_thrust_idata.begin(), dev_thrust_idata.end(), dev_thrust_odata.begin());

            timer().endGpuTimer();

            int *dev_odata = thrust::raw_pointer_cast(dev_thrust_odata.data());
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy back failed!");
        }
    }
}

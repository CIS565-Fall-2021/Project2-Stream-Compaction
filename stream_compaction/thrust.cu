#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

#define checkCUDAErrorWithLine(msg) checkCUDAErrorFn(msg, __FILE__, __LINE__)
#define blockSize 128

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int N, int *odata, const int *idata) {

        	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

						// copy data to gpu buffer
						int* dev_odata;
						int* dev_idata;
						hipMalloc((void**)&dev_odata, N * sizeof(int));
						checkCUDAErrorWithLine("hipMalloc dev_odata failed!");
						hipMalloc((void**)&dev_idata, N * sizeof(int));
						checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

						hipMemcpy(dev_odata, odata, sizeof(int) * N, hipMemcpyHostToDevice);
						hipMemcpy(dev_idata, idata, sizeof(int) * N, hipMemcpyHostToDevice);


						timer().startGpuTimer();

						thrust::device_ptr<int> dev_thrust_odata(dev_odata);
						thrust::device_ptr<int> dev_thrust_idata(dev_idata);

						 // TODO use `thrust::exclusive_scan`
						            // example: for device_vectors dv_in and dv_out:
						            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
						thrust::inclusive_scan(dev_thrust_idata, dev_thrust_idata+N, dev_thrust_odata);

						//hipDeviceSynchronize();
						timer().endGpuTimer();

						hipMemcpy(odata, dev_odata, sizeof(int) * N, hipMemcpyDeviceToHost);

						hipDeviceSynchronize();
						hipFree(dev_odata);
						hipFree(dev_idata);

        }
    }
}

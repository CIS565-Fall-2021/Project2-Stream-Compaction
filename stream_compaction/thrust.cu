#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            thrust::device_vector<int> dev_odata(odata, odata + n);
            thrust::device_vector<int> dev_idata(idata, idata + n);

            timer().startGpuTimer();
            thrust::exclusive_scan(dev_idata.begin(), dev_idata.end(), dev_odata.begin());
            
            timer().endGpuTimer();

            thrust::copy(dev_odata.begin(), dev_odata.end(), odata);

        }

        void sort(int n, int *odata, const int *idata) {

            std::cout << std::endl;
            thrust::copy(idata, idata + n, odata);

            timer().startGpuTimer();
            thrust::sort(odata, odata+n);
            
            timer().endGpuTimer();

        }
    }
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_in;
            hipMalloc((void**)&dev_in, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_in failed!");

            int* dev_out;
            hipMalloc((void**)&dev_out, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_out failed!");

            hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to device failed!");

            thrust::device_ptr<int> dev_thrust_in = thrust::device_pointer_cast(dev_in);
            thrust::device_ptr<int> dev_thrust_out = thrust::device_pointer_cast(dev_out);

            timer().startGpuTimer();

            thrust::exclusive_scan(dev_thrust_in, dev_thrust_in + n, dev_thrust_out);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from device failed!");

            hipFree(dev_in);
            checkCUDAError("hipFree dev_in failed!");

            hipFree(dev_out);
            checkCUDAError("hipFree dev_out failed!");
        }
    }
}

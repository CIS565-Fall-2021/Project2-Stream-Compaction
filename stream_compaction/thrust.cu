#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {
	using StreamCompaction::Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}
	/**
	 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
	 */
	void scan(int n, int *odata, const int *idata) {
		thrust::host_vector<int> host_idata(n);
		std::copy(idata, idata + n, host_idata.data()); //thrust::copy didn't work with raw pointer
		thrust::device_vector<int> dv_in = host_idata;
		thrust::device_vector<int> dv_out(n);

		timer().startGpuTimer();
		thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
		timer().endGpuTimer();

		thrust::copy(dv_out.begin(), dv_out.end(), odata);
	}
}
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#include <iostream> // testing 
#include <cassert> // for assert()



namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void convertFromInclusiveToExclusive(const int* inputArray,
            int* outputArray, int inputSize)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            // convert inclusive scan into exclusive scan by shifting 
            // all elements to the right by one position and fill the frist 
            // element and out-of-bound elements with 0. 
            if (i < inputSize && i != 0)
            {
        
                outputArray[i] = inputArray[i - 1];
            }
            else {
                outputArray[i] = 0;
            }
        }

        __device__ void computeScanToOutputArray(const int* inputArray, int* outputArray,
            int* XY, int inputSize)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            if (i < inputSize)
            {
                XY[threadIdx.x] = inputArray[i];
            }
            else {
                XY[threadIdx.x] = 0;
            }
            // perform naive scan
            for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
            {
                // make sure that input is in place
                __syncthreads();
                int previousValue = 0;
                int previousIndex = threadIdx.x - stride;
                if (previousIndex >= 0)
                {
                    previousValue = XY[previousIndex];
                }
                int temp = XY[threadIdx.x] + previousValue;
                // make sure previous output has been consumed
                __syncthreads();
                XY[threadIdx.x] = temp;
            }

            // each thread writes its result into the output array
            outputArray[i] = XY[threadIdx.x];
        }
        
        __global__ void kernNaiveGPUScanFirstStep(const int* inputArray, 
            int* outputArray, int* SumArray, int inputSize)
        {
            // Each thread loads one value from the input array into shared 
            // memory array XY
            __shared__ int XY[sectionSize];
            computeScanToOutputArray(inputArray, outputArray, XY, inputSize);

            // the last thread in the block should write the output value of 
            // the last XY element in the block to the blockIdx.x position of 
            // SumArray

            // make sure XY[sectionSize - 1] has the correct partial sum
            __syncthreads(); 
            if (threadIdx.x == blockDim.x - 1)
            {
                SumArray[blockIdx.x] = XY[sectionSize - 1];
            }
        }

        __global__ void kernNaiveGPUScanSecondStep(const int* inputArray, 
            int* outputArray, int inputSize)
        {
            // Each thread loads one value from the input array into shared 
            // memory array XY
            __shared__ int XY[MAX_SUM_ARRAY_SIZE];
            computeScanToOutputArray(inputArray, outputArray, XY, inputSize);
        }

        __global__ void kernNaiveGPUScanThirdStep(const int* inputArray, 
            int* outputArray, int inputSize)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            if (i < inputSize && blockIdx.x > 0)
            {
                outputArray[i] += inputArray[blockIdx.x - 1];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int size = n * sizeof(int);
            int sumArrayNumEle = (n + blockSize - 1) / blockSize;
            assert(sumArrayNumEle <= 1024 && "Sum Array has more than 1024 elements!");
            int sumArraySize = sumArrayNumEle * sizeof(int);

            int* d_InputData;
            int* d_OutputData;
            int* d_OutputExclusiveData;
            int* d_SumArray;
            int* d_SumArrayOutput;
            int* d_SumArrayAx;

            hipMalloc((void**)&d_InputData, size);
            checkCUDAError("hipMalloc d_InputData failed!");

            hipMalloc((void**)&d_OutputData, size);
            checkCUDAError("hipMalloc d_OutputData failed!");

            hipMalloc((void**)&d_OutputExclusiveData, size);
            checkCUDAError("hipMalloc d_OutputExclusiveData failed!");

            hipMalloc((void**)&d_SumArray, sumArraySize);
            checkCUDAError("hipMalloc d_SumArray failed!");

            hipMalloc((void**)&d_SumArrayOutput, sumArraySize);
            checkCUDAError("hipMalloc d_SumArrayOutput failed!");

            hipMalloc((void**)&d_SumArrayAx, sumArraySize);
            checkCUDAError("hipMalloc d_SumArrayOutput failed!");

            hipMemcpy(d_InputData, idata, size, hipMemcpyHostToDevice);

            dim3 dimGridArray((n + blockSize - 1) / blockSize, 1, 1);
            dim3 dimBlockArray(blockSize, 1, 1);

            
            dim3 dimGridSumArray(1, 1, 1);
            dim3 dimBlockSumArray(sumArrayNumEle, 1, 1);

            // for testing
            int* sumArray = new int[sumArrayNumEle];
            int* sumArrayOutput = new int[sumArrayNumEle];

            timer().startGpuTimer();
            // First step: compute the scan result for individual sections
            // then, store their block sum to sumArray
            kernNaiveGPUScanFirstStep << <dimGridArray, dimBlockArray >> > (d_InputData,
                d_OutputData, d_SumArray, n);
            checkCUDAError("kernNaiveGPUScanFirstStep failed!");
#if 0
            hipDeviceSynchronize();
            hipMemcpy(odata, d_OutputData, size, hipMemcpyDeviceToHost);
            checkCUDAError("memCpy back failed!");

            hipMemcpy(sumArray, d_SumArray, sumArraySize, hipMemcpyDeviceToHost);
            checkCUDAError("memCpy back failed!");

            std::cout << '\n';
            for (int i = 0; i < n; i++)
            {
                std::cout << odata[i] << ' ';
                if ((i + 1) % 8 == 0) {
                    std::cout << std::endl;
                }
            }

            std::cout << '\n';
            for (int i = 0; i < sumArrayNumEle; i++)
            {
                std::cout << sumArray[i] << ' ';
            }

            std::cout << '\n';
#endif
            // Second step: scan block sums
            kernNaiveGPUScanSecondStep << <dimGridSumArray, dimBlockSumArray >> > (
                d_SumArray, d_SumArrayOutput, sumArrayNumEle);
            checkCUDAError("kernNaiveGPUScanSecondStep failed!");
#if 0

            hipMemcpy(sumArrayOutput, d_SumArrayOutput, sumArraySize,
                hipMemcpyDeviceToHost);
            checkCUDAError("memCpy back failed!");

            printf("\n");

            for (int i = 0; i < sumArrayNumEle; i++)
            {
                std::cout << sumArrayOutput[i] << ' ';
            }

            printf("\n");

#endif
            // Third step: add scanned block sum i to all values of scanned block
            // i + 1
            kernNaiveGPUScanThirdStep << <dimGridArray, dimBlockArray >> > (
                d_SumArrayOutput, d_OutputData, n);
            checkCUDAError("kernNaiveGPUScanThirdStep failed!");

           // hipDeviceSynchronize();

            // Last step:

            convertFromInclusiveToExclusive << <dimGridArray, dimBlockArray >> > (
                d_OutputData, d_OutputExclusiveData, n);
            checkCUDAError("convertFromInclusiveToExclusive failed!");

            timer().endGpuTimer();

            hipMemcpy(odata, d_OutputExclusiveData, size, hipMemcpyDeviceToHost);
            checkCUDAError("memCpy back failed!");

            // cleanup
            hipFree(d_InputData);
            hipFree(d_OutputData);
            hipFree(d_OutputExclusiveData);
            hipFree(d_SumArray);
            hipFree(d_SumArrayOutput);
            checkCUDAError("hipFree failed!");
        }
    }
}

#if 0
void unitTestConversion()
{
    // for testing
    int numObject = 8;
    int size = numObject * sizeof(int);
    int* toyExclusiveArray = new int[numObject];
    int* toyInclusiveArray = new int[numObject] {3, 4, 11, 11, 15, 16, 22, 25};

    int* dev_toyExclusiveArray;
    int* dev_toyInclusiveArray;

    hipMalloc((void**)&dev_toyExclusiveArray, size);
    checkCUDAError("hipMalloc dev_toyExclusiveArray failed!");

    hipMalloc((void**)&dev_toyInclusiveArray, size);
    checkCUDAError("hipMalloc dev_toyInclusiveArray failed!");

    hipMemcpy(dev_toyInclusiveArray, toyInclusiveArray, size,
        hipMemcpyHostToDevice);

    dim3 dimGridArray((numObject + blockSize - 1) / blockSize, 1, 1);
    dim3 dimBlockArray(blockSize, 1, 1);
    convertFromInclusiveToExclusive << <dimGridArray, dimBlockArray >> > (
        dev_toyInclusiveArray, dev_toyExclusiveArray, numObject);

    hipMemcpy(toyExclusiveArray, dev_toyExclusiveArray, size,
        hipMemcpyDeviceToHost);
    checkCUDAError("memCpy back failed!");

    printf("\n");

    for (int i = 0; i < numObject; i++)
    {
        std::cout << toyExclusiveArray[i] << '\n';
    }

    printf("\n");

}
#endif
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include ""
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernScanNaive(int n, int depth, const int *dev_src, int *dev_dest) {
            int index = threadIdx.x + (blockDim.x * blockIdx.x);
            if (index >= n) return;

            int depthPow = 1 << (depth - 1);
            if (index >= depthPow) { // update curr
                dev_dest[index] = dev_src[index - depthPow] + dev_src[index];
            }
            else { // update from previous
                dev_dest[index] = dev_src[index];
            }
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_A;
            int* dev_B;
            hipMalloc((void**)&dev_A, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc failed on dev_A!");
            hipMalloc((void**)&dev_B, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc failed on dev_B!");

            hipMemcpy(dev_A, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(dev_B, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            int maxDepth = ilog2ceil(n);
            for (int d = 1; d <= maxDepth; d++) {
                kernScanNaive << <fullBlocksPerGrid, blockSize >> > (n, d, dev_A, dev_B);
                std::swap(dev_A, dev_B);
            }
            timer().endGpuTimer();

            hipMemcpy(odata + 1, dev_A, (n - 1) * sizeof(int), hipMemcpyDeviceToHost); // copy over with shift for exclusive scan
            odata[0] = 0; // set ident

            hipFree(dev_A);
            checkCUDAErrorFn("hipFree failed on dev_A!");
            hipFree(dev_B);
            checkCUDAErrorFn("hipFree failed on dev_B!");
        }
    }
}

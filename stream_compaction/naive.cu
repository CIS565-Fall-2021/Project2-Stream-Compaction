#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <vector>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernNaiveScanStep(int n, int offset, int* odata, const int* idata) {
          int index = threadIdx.x + (blockIdx.x * blockDim.x);
          if (index >= 0 && index < n) {
            if (index >= offset) {
              odata[index] = idata[index - offset] + idata[index];
            }
            else {
              odata[index] = idata[index];
            }
          }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            dim3 fullBlocksPerGrid = ((n + blockSize - 1) / blockSize);

            int* dev_buf0;
            hipMalloc((void**)&dev_buf0, n * sizeof(int));
            checkCUDAError("hipMalloc dev_buf0 failed!");

            int* dev_buf1;
            hipMalloc((void**)&dev_buf1, n * sizeof(int));
            checkCUDAError("hipMalloc dev_buf0 failed!");
            
            hipMemcpy(dev_buf0, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to device failed!");

            for (int offset = 1; offset < n; offset <<= 1) {
              kernNaiveScanStep << <fullBlocksPerGrid, blockSize >> > (n, offset, dev_buf1, dev_buf0);
              checkCUDAError("kernNaiveScanStep failed!");

              std::swap(dev_buf0, dev_buf1);
            }

            hipMemcpy(&odata[1], dev_buf0, sizeof(int) * (n - 1), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from device failed!");

            std::vector<int> odataView(n);
            memcpy(odataView.data(), odata, sizeof(int) * n);

            std::vector<int> idataView(n);
            memcpy(idataView.data(), idata, sizeof(int) * n);

            hipFree(dev_buf0);
            checkCUDAError("hipFree dev_buf0 failed!");

            hipFree(dev_buf1);
            checkCUDAError("hipFree dev_buf1 failed!");

            timer().endGpuTimer();
        }
    }
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#include <iostream> // testing 

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // write a kernel to convert from inclusive scan to exclusive scan

        __global__ void convertFromInclusiveToExclusive(const int* inputArray,
            int* outputArray, int inputSize)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            // convert inclusive scan into exclusive scan by shifting 
            // all elements to the right by one position and fill the frist 
            // element and out-of-bound elements with 0. 
            if (i < inputSize && i != 0)
            {
        
                outputArray[i] = inputArray[i - 1];
            }
            else {
                outputArray[i] = 0;
            }
        }

        void unitTestConversion()
        {
            // for testing
            int numObject = 8;
            int size = numObject * sizeof(int);
            int* toyExclusiveArray = new int[numObject];
            int* toyInclusiveArray = new int[numObject] {3, 4, 11, 11, 15, 16, 22, 25};

            int* dev_toyExclusiveArray;
            int* dev_toyInclusiveArray;

            hipMalloc((void**)&dev_toyExclusiveArray, size);
            checkCUDAError("hipMalloc dev_toyExclusiveArray failed!");

            hipMalloc((void**)&dev_toyInclusiveArray, size);
            checkCUDAError("hipMalloc dev_toyInclusiveArray failed!");

            hipMemcpy(dev_toyInclusiveArray, toyInclusiveArray, size,
                hipMemcpyHostToDevice);
 
            dim3 dimGridArray((numObject + blockSize - 1) / blockSize, 1, 1);
            dim3 dimBlockArray(blockSize, 1, 1);
            convertFromInclusiveToExclusive <<<dimGridArray, dimBlockArray >>> (
                dev_toyInclusiveArray, dev_toyExclusiveArray, numObject);

            hipMemcpy(toyExclusiveArray, dev_toyExclusiveArray, size,
                hipMemcpyDeviceToHost);
            checkCUDAError("memCpy back failed!");

            printf("\n");

            for (int i = 0; i < numObject; i++)
            {
                std::cout << toyExclusiveArray[i] << '\n';
            }

            printf("\n");

        }

        __device__ void computeScanToOutputArray(const int* inputArray, int* outputArray,
            int* XY, int inputSize)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            if (i < inputSize)
            {
                XY[threadIdx.x] = inputArray[i];
            }
            else {
                XY[threadIdx.x] = 0;
            }
            // perform naive scan
            for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
            {
                // make sure that input is in place
                __syncthreads();
                int previousValue = 0;
                int previousIndex = threadIdx.x - stride;
                if (previousIndex >= 0)
                {
                    previousValue = XY[previousIndex];
                }
                int temp = XY[threadIdx.x] + previousValue;
                // make sure previous output has been consumed
                __syncthreads();
                XY[threadIdx.x] = temp;
            }

            // each thread writes its result into the output array
            outputArray[i] = XY[threadIdx.x];
        }
        
        __global__ void kernNaiveGPUScanFirstStep(const int* inputArray, 
            int* outputArray, int* SumArray, int inputSize)
        {
            // Each thread loads one value from the input array into shared 
            // memory array XY
            __shared__ int XY[sectionSize];
            computeScanToOutputArray(inputArray, outputArray, XY, inputSize);

            // the last thread in the block should write the output value of 
            // the last XY element in the block to the blockIdx.x position of 
            // SumArray

            // make sure XY[sectionSize - 1] has the correct partial sum
            __syncthreads(); 
            if (threadIdx.x == blockDim.x - 1)
            {
                SumArray[blockIdx.x] = XY[sectionSize - 1];
            }
        }

        __global__ void kernNaiveGPUScanSecondStep(const int* inputArray, 
            int* outputArray, int inputSize)
        {
            // Each thread loads one value from the input array into shared 
            // memory array XY
            __shared__ int XY[sectionSize];
            computeScanToOutputArray(inputArray, outputArray, XY, inputSize);
        }

        __global__ void kernNaiveGPUScanThirdStep(const int* inputArray, 
            int* outputArray, int inputSize)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            if (i < inputSize && blockIdx.x > 0)
            {
                outputArray[i] += inputArray[blockIdx.x - 1];
            }
        }

        

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int size = n * sizeof(int);
            int sumArrayNumEle = (n + blockSize - 1) / blockSize;
            int sumArraySize = sumArrayNumEle * sizeof(int);

            int* d_InputData;
            int* d_OutputData;
            int* d_OutputExclusiveData;
            int* d_SumArray;
            int* d_SumArrayOutput;
            int* d_SumArrayAx;

            hipMalloc((void**)&d_InputData, size);
            checkCUDAError("hipMalloc d_InputData failed!");

            hipMalloc((void**)&d_OutputData, size);
            checkCUDAError("hipMalloc d_OutputData failed!");

            hipMalloc((void**)&d_OutputExclusiveData, size);
            checkCUDAError("hipMalloc d_OutputExclusiveData failed!");

            hipMalloc((void**)&d_SumArray, sumArraySize);
            checkCUDAError("hipMalloc d_SumArray failed!");

            hipMalloc((void**)&d_SumArrayOutput, sumArraySize);
            checkCUDAError("hipMalloc d_SumArrayOutput failed!");

            hipMalloc((void**)&d_SumArrayAx, sumArraySize);
            checkCUDAError("hipMalloc d_SumArrayOutput failed!");

            hipMemcpy(d_InputData, idata, size, hipMemcpyHostToDevice);

            dim3 dimGridArray((n + blockSize - 1) / blockSize, 1, 1);
            dim3 dimBlockArray(blockSize, 1, 1);

            dim3 dimGridSumArray((sumArrayNumEle + blockSize - 1) / blockSize, 1, 1);
            dim3 dimBlockSumArray(blockSize, 1, 1);

            // for testing
            int* sumArray = new int[sumArrayNumEle];
            int* sumArrayOutput = new int[sumArrayNumEle];

            timer().startGpuTimer();
            // First step: compute the scan result for individual sections
            // then, store their block sum to sumArray
            kernNaiveGPUScanFirstStep << <dimGridArray, dimBlockArray >> > (d_InputData,
                d_OutputData, d_SumArray, n);
            checkCUDAError("kernNaiveGPUScanFirstStep failed!");

            // hipDeviceSynchronize();

            hipMemcpy(odata, d_OutputData, size, hipMemcpyDeviceToHost);
            checkCUDAError("memCpy back failed!");

            hipMemcpy(sumArray, d_SumArray, sumArraySize, hipMemcpyDeviceToHost);
            checkCUDAError("memCpy back failed!");

            std::cout << '\n';
            for (int i = 0; i < n; i++)
            {
                std::cout << odata[i] << ' ';
                if ((i + 1) % 8 == 0) {
                    std::cout << std::endl;
                }
            }

            std::cout << '\n';
            for (int i = 0; i < sumArrayNumEle; i++)
            {
                std::cout << sumArray[i] << ' ';
            }

            std::cout << '\n';

            // Second step: scan block sums
            kernNaiveGPUScanFirstStep << <dimGridSumArray, dimBlockSumArray >> > (d_SumArray,
                d_SumArrayOutput, d_SumArrayAx, n);

            kernNaiveGPUScanThirdStep << <dimGridSumArray, dimBlockSumArray >> > (
                d_SumArrayAx, d_SumArrayOutput, n);

            hipMemcpy(sumArrayOutput, d_SumArrayOutput, sumArraySize,
                hipMemcpyDeviceToHost);
            checkCUDAError("memCpy back failed!");

            printf("\n");

            for (int i = 0; i < sumArrayNumEle; i++)
            {
                std::cout << sumArrayOutput[i] << ' ';
            }

            printf("\n");
            //hipDeviceSynchronize();

            // Third step: add scanned block sum i to all values of scanned block
            // i + 1
            kernNaiveGPUScanThirdStep << <dimGridArray, dimBlockArray >> > (
                d_SumArrayOutput, d_OutputData, n);
            checkCUDAError("kernNaiveGPUScanThirdStep failed!");

            // Last step:
            convertFromInclusiveToExclusive << <dimGridArray, dimBlockArray >> > (
                d_OutputData, d_OutputExclusiveData, n);
            checkCUDAError("convertFromInclusiveToExclusive failed!");

            timer().endGpuTimer();

            hipMemcpy(odata, d_OutputExclusiveData, size, hipMemcpyDeviceToHost);
            checkCUDAError("memCpy back failed!");

            // cleanup
            hipFree(d_InputData);
            hipFree(d_OutputData);
            hipFree(d_OutputExclusiveData);
            hipFree(d_SumArray);
            hipFree(d_SumArrayOutput);
            checkCUDAError("hipFree failed!");
        }
    }
}

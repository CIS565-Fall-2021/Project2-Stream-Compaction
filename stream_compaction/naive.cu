#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <iostream>

// nvcc does not seem to like variadic macros, so we have to define
// one for each kernel parameter list:

#define blockSize 128

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernel_naive_parallel_scan(const int* read, int* write, int d, int n)
        {
          int k = blockIdx.x * blockDim.x + threadIdx.x;

          if (k > n - 1)
            return;

          int step = (int)(powf(2, d - 1) + 0.5);
          if (k >= step)
            write[k] = read[k - step] + read[k];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // allocate memory for the read write buffers
            int* devRead, int* devWrite;
            hipMalloc((void**)&devRead, n * sizeof(int));
            hipMalloc((void**)&devWrite, n * sizeof(int));

            // Copy idata to read and write buffer
            hipMemcpy(devRead, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(devWrite, devRead, n * sizeof(int), hipMemcpyDeviceToDevice);
            
            // define kernel dimension
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            // run naive scan
            for (int d = 1; d <= ilog2ceil(n); d++)
            {
              kernel_naive_parallel_scan<<<fullBlocksPerGrid, blockSize>>>(devRead, devWrite, d, n);

              // swap read and write
              hipMemcpy(devRead, devWrite, n * sizeof(int), hipMemcpyDeviceToDevice); // TODO: is there another way?
            }

            // Copy write buffer to odata
            odata[0] = 0;
            hipMemcpy(odata + 1, devRead, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);

            // free memory
            hipFree(devRead);
            hipFree(devWrite);
            timer().endGpuTimer();
        }
    }
}

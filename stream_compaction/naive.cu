#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction
{
    namespace Naive
    {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer &timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernScanNaive(int n, int layer, int offset, int *odata, const int *idata)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n)
            {
                return;
            }
            int tmp = idata[index];
            odata[index] = tmp + (index >= offset) * idata[index - offset];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata)
        {
            int power = ilog2ceil(n);
            int size = pow(2, power);
            int offset = size - n;
            dim3 fullBlocksPerGrid((size + blockSize - 1) / blockSize);
            int *bufA;
            int *bufB;
            hipMalloc((void **)&bufA, size * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc bufA failed!");
            hipMalloc((void **)&bufB, size * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc bufB failed!");

            hipMemset(bufA, 0, size * sizeof(int));
            hipMemset(bufB, 0, size * sizeof(int));
            hipMemcpy(bufA + offset, idata, n * sizeof(int), hipMemcpyHostToDevice);
            int *tmp;
            timer().startGpuTimer();
            // TODO
            for (int layer = 0; layer < power; layer++)
            {
                // invoke kernel
                int offset = pow(2, layer);
                kernScanNaive<<<fullBlocksPerGrid, blockSize>>>(size, layer, offset, bufB, bufA);
                hipDeviceSynchronize();
                // swap bufA and bufB
                tmp = bufA;
                bufA = bufB;
                bufB = tmp;
            }
            hipDeviceSynchronize();
            timer().endGpuTimer();
            hipMemcpy(odata + 1, bufA + offset, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
            odata[0] = 0;
            hipFree(bufA);
            checkCUDAErrorWithLine("hipFree bufA failed!");
            hipFree(bufB);
            checkCUDAErrorWithLine("hipFree bufB failed!");
        }
    }
}

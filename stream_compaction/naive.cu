#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            int offset, *dev_odata, *dev_idata;
            // malloc memory before timing
            hipMalloc((void**)&dev_odata, n * sizeof(int));
	        hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            timer().startGpuTimer();
            // calling kernel function in for loop, will be executed in parallel
            for (int d=1; d<ilog2ceil(n);d++){
                offset = 1 << (d - 1);
		        kernScan << <fullBlocksPerGrid, blockSize>> >(offset, n, dev_odata, dev_idata);
                // ping pong buffer
		        std::swap(dev_odata, dev_idata);
            }
            timer().endGpuTimer();
            printf("Naive scan: %f ms\n", timer().getGpuElapsedTimeForPreviousOperation());
            hipMemcpy(odata + 1, dev_odata, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
	        odata[0] = 0;
            // free memory
            hipFree(dev_odata);
	        hipFree(dev_idata);
        }

        __global__ void kernScan(int offset, int n, int *dev_odata, int *dev_idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) return;
            if (index >= offset) {
                dev_odata[index] = dev_idata[index - offset] + dev_idata[index];
            }
            else {
                dev_odata[index] = dev_idata[index];
            }
        }       
    }
}
